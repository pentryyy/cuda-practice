#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    const int numElements = 1024;
    size_t size = numElements * sizeof(float);

    // Выделение памяти на хосте
    float* h_A = new float[numElements];
    float* h_B = new float[numElements];
    float* h_C = new float[numElements];

    // Инициализация данных
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Выделение памяти на устройстве
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Копирование данных на устройство
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Запуск ядра
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Копирование результатов обратно
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Проверка результатов
    bool error = false;
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
            error = true;
            break;
        }
    }
    
    std::cout << (error ? "Error!" : "Success!") << std::endl;

    // Освобождение памяти
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}