#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 2048
#define BLOCK_SIZE 32

__global__ void matrixMul(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    if (row < N && col < N) {
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    
    size_t size = N * N * sizeof(float);

    // Выделение памяти на хосте
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    printf("[INFO] Host memory allocated.\n");

    // Инициализация данных
    printf("[INFO] Initializing matrices...\n");
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (float)(i % N);
        h_B[i] = (float)(i / N);
    }
    printf("[INFO] Data initialized.\n");

    // Выделение памяти на устройстве
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    printf("[INFO] Device memory allocated.\n");

    // Копирование данных на устройство
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    printf("[INFO] Data copied to device.\n");

    // Запуск ядра
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / BLOCK_SIZE, N / BLOCK_SIZE);
    printf("[INFO] Launching kernel with %dx%d blocks and %dx%d threads.\n", blocks.x, blocks.y, threads.x, threads.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMul<<<blocks, threads>>>(d_A, d_B, d_C);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpuTime;
    hipEventElapsedTime(&gpuTime, start, stop);
    printf("[INFO] GPU calculation time: %.2f ms\n", gpuTime);

    // Копирование результатов обратно
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    printf("[INFO] Results copied to host.\n");

    // Освобождение памяти
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    printf("[INFO] Memory freed.\n");

    return 0;
}
