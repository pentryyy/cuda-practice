#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void complexKernel(float *dA, float *dB, float *dC, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        float ab = dA[i] * dB[i];
        float sum = 0.0f;
        for (int j = 0; j < 100; j++) {
            sum += sinf(ab + j);
        }
        dC[i] = sum;
    }
}

int main() {
    const int N = 512 * 50000;
    const int mem_size = N * sizeof(float);

    float *hA, *hB, *hC;
    float *dA, *dB, *dC;

    hipHostAlloc((void**)&hA, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hB, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hC, mem_size, hipHostMallocDefault);

    // Инициализация данных
    for (int i = 0; i < N; i++) {
        hA[i] = sinf(i);
        hB[i] = cosf(2 * i - 5);
    }
    printf("[INFO] Initializing data...\n");

    hipMalloc((void**)&dA, mem_size);
    hipMalloc((void**)&dB, mem_size);
    hipMalloc((void**)&dC, mem_size);

    hipMemcpy(dA, hA, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, mem_size, hipMemcpyHostToDevice);

    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    printf("[INFO] Kernel launched.\n");
    complexKernel<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);

    printf("[INFO] Copying results...\n");
    hipMemcpy(hC, dC, mem_size, hipMemcpyDeviceToHost);

    // Освобождение памяти
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    printf("[INFO] Memory and streams released.\n");

    return 0;
}
