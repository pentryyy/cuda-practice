#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *dA, float *dB, float *dC, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) dC[i] = dA[i] + dB[i];
}

int main() {
    const int N = 512 * 50000;
    const int mem_size = N * sizeof(float);

    float *hA, *hB, *hC;
    float *dA, *dB, *dC;

    // Использование pinned-памяти
    hipHostAlloc((void**)&hA, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hB, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hC, mem_size, hipHostMallocDefault);

    // Инициализация данных
    printf("[INFO] Initializing data...\n");
    for (int i = 0; i < N; i++) {
        hA[i] = 1.0f / ((i + 1.0f) * (i + 1.0f));
        hB[i] = expf(1.0f / (i + 1.0f));
    }
    printf("[INFO] Data initialized.\n");

    hipMalloc((void**)&dA, mem_size);
    hipMalloc((void**)&dB, mem_size);
    hipMalloc((void**)&dC, mem_size);

    hipMemcpy(dA, hA, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, mem_size, hipMemcpyHostToDevice);

    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    printf("[INFO] Kernel launched.\n");
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(dA, dB, dC, N);

    printf("[INFO] Copying results...\n");
    hipMemcpy(hC, dC, mem_size, hipMemcpyDeviceToHost);

    // Освобождение pinned-памяти
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    printf("[INFO] Memory and streams released.\n");

    return 0;
}
