#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *dA, float *dB, float *dC, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) dC[i] = dA[i] + dB[i];
}

int main() {
    const int N = 512 * 50000;
    const int mem_size = N * sizeof(float);
    const int num_streams = 4;

    hipStream_t streams[num_streams];

    float *hA, *hB, *hC;
    float *dA, *dB, *dC;

    // Выделение pinned-памяти
    hipHostAlloc((void**)&hA, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hB, mem_size, hipHostMallocDefault);
    hipHostAlloc((void**)&hC, mem_size, hipHostMallocDefault);
    printf("[INFO] Host memory allocated.\n");

    // Инициализация данных
    printf("[INFO] Initializing data...\n");
    for (int i = 0; i < N; i++) {
        hA[i] = 1.0f / ((i + 1.0f) * (i + 1.0f));
        hB[i] = expf(1.0f / (i + 1.0f));
    }
    printf("[INFO] Data initialized on host.\n");

    hipMalloc((void**)&dA, mem_size);
    hipMalloc((void**)&dB, mem_size);
    hipMalloc((void**)&dC, mem_size);
    printf("[INFO] Device memory allocated.\n");

    // Создание потоков
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
    }

    // Асинхронные операции
    int chunk_size = N / num_streams;
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        hipMemcpyAsync(dA + offset, hA + offset, chunk_size * sizeof(float),
                        hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(dB + offset, hB + offset, chunk_size * sizeof(float),
                        hipMemcpyHostToDevice, streams[i]);
    }

    int threadsPerBlock = 512;
    int blocksPerGrid = (chunk_size + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>(dA + offset, dB + offset, dC + offset, chunk_size);
    }

    printf("[INFO] Copying results to host...\n");
    for (int i = 0; i < num_streams; i++) {
        int offset = i * chunk_size;
        hipMemcpyAsync(hC + offset, dC + offset, chunk_size * sizeof(float),
                        hipMemcpyDeviceToHost, streams[i]);
    }

    // Синхронизация
    hipDeviceSynchronize();
    printf("[INFO] All operations completed.\n");

    // Освобождение ресурсов
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    for (int i = 0; i < num_streams; i++) {
        hipStreamDestroy(streams[i]);
    }
    printf("[INFO] Memory freed.\n");

    return 0;
}
