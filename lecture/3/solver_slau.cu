#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define EPS 1e-15
#define N 10240

// Ядро для решения СЛАУ методом итераций
__global__ void Solve(double *dA, double *dF, double *dX0, double *dX1, int size) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < size) {
        double aa, sum = 0.0;
        for (int j = 0; j < size; j++) {
            sum += dA[j + t * size] * dX0[j];
            if (j == t) aa = dA[j + t * size];
        }
        dX1[t] = dX0[t] + (dF[t] - sum) / aa;
    }
}

// Ядро для вычисления погрешности
__global__ void Eps(double *dX0, double *dX1, double *delta, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        delta[i] = fabs(dX0[i] - dX1[i]);
        dX0[i] = dX1[i];
    }
}

int main() {
    double *hA, *hF, *hX0, *hX1, *hDelta;
    double *dA, *dF, *dX0, *dX1, *delta;
    
    const int mem_sizeA = N * N * sizeof(double);
    const int mem_sizeX = N * sizeof(double);
    const int threadsPerBlock = 512;
    const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Выделение памяти на хосте
    hA = (double*)malloc(mem_sizeA);
    hF = (double*)malloc(mem_sizeX);
    hX0 = (double*)malloc(mem_sizeX);
    hX1 = (double*)malloc(mem_sizeX);
    hDelta = (double*)malloc(mem_sizeX);
    printf("[INFO] Host memory allocated.\n");

    // Инициализация данных (примерная)
    printf("[INFO] Initializing data...\n");
    for (int i = 0; i < N; i++) {
        hF[i] = 1.0;
        hX0[i] = 0.0;
        for (int j = 0; j < N; j++) {
            hA[j + i * N] = (i == j) ? 2.0 : 0.1; // Диагональное преобладание
        }
    }
    printf("[INFO] Data initialized.\n");

    // Выделение памяти на устройстве
    hipMalloc((void**)&dA, mem_sizeA);
    hipMalloc((void**)&dF, mem_sizeX);
    hipMalloc((void**)&dX0, mem_sizeX);
    hipMalloc((void**)&dX1, mem_sizeX);
    hipMalloc((void**)&delta, mem_sizeX);
    printf("[INFO] Device memory allocated.\n");

    // Копирование данных на устройство
    hipMemcpy(dA, hA, mem_sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dF, hF, mem_sizeX, hipMemcpyHostToDevice);
    hipMemcpy(dX0, hX0, mem_sizeX, hipMemcpyHostToDevice);
    printf("[INFO] Data copied to device.\n");

    // Итерационный процесс
    double eps = 1.0;
    int iter = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    printf("[INFO] Starting iterations...\n");
    while (eps > EPS) {
        Solve<<<blocksPerGrid, threadsPerBlock>>>(dA, dF, dX0, dX1, N);
        Eps<<<blocksPerGrid, threadsPerBlock>>>(dX0, dX1, delta, N);
        hipMemcpy(hDelta, delta, mem_sizeX, hipMemcpyDeviceToHost);

        eps = 0.0;
        for (int i = 0; i < N; i++) eps += hDelta[i];
        eps /= N;
        iter++;
        printf("[INFO] Iteration %d: eps = %e\n", iter, eps);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float timerValueGPU;
    hipEventElapsedTime(&timerValueGPU, start, stop);
    printf("[INFO] GPU calculation time: %.2f ms\n", timerValueGPU);

    // Освобождение памяти
    free(hA);
    free(hF);
    free(hX0);
    free(hX1);
    free(hDelta);
    hipFree(dA);
    hipFree(dF);
    hipFree(dX0);
    hipFree(dX1);
    hipFree(delta);
    printf("[INFO] Memory freed.\n");

    return 0;
}
